
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define MAX_SIZE 6
#define VARIATION 93
#define FIRST 33
#define SPREAD 4

//Holds the password in constant memory
__constant__ char password[MAX_SIZE];

//Checks the tested password against the actual password
__device__ bool checkWord(char* guessword, int length){
  bool checkMatch = true;
  for(int i = 0; i <= length + 1; i++){
    if(guessword[i] != password[i]){
      checkMatch = false;
      break;
    }
  }
  return checkMatch;
}

//Creates a password based on the blockId
__device__ void wordStarter(char* guessword, int length){
  guessword[0] = FIRST + threadIdx.x;
  if(length > 4) length = 4;
  switch(length){
    case 4: guessword[4] = FIRST + (blockIdx.y / VARIATION);
    case 3: guessword[3] = FIRST + (blockIdx.y % VARIATION);
    case 2: guessword[2] = FIRST + (blockIdx.x / VARIATION);
    case 1: guessword[1] = FIRST + (blockIdx.x % VARIATION);
  }
}

//Main kernel function; searches for a suitable password
__global__ void startCrackin(char* testword){
  int length = 0;
  if(blockIdx.y > 0){
    if(blockIdx.y > VARIATION){
      length = 4;
    } else {
      length = 3;
    }
  } else if (blockIdx.x > 0){
    if(blockIdx.x > VARIATION){
      length = 2;
    } else {
      length = 1;
    }
  }
  char guessword[MAX_SIZE];
  guessword[length] = 0;
  wordStarter(guessword, length);
  if(checkWord(guessword, length)){
    for(int i = 0; i <= length + 1; i++){
      testword[i] = guessword[i];
    }
  }
}

//Main function; runs password cracker and timer to determine speed
int main(void){
  char* testword;
  char* hostPassword;
  char* cudaTestword;
  int starters = VARIATION * VARIATION;
  dim3 blocks = dim3(starters, starters, 1);
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  testword = (char*)malloc(MAX_SIZE * sizeof(char));
  hostPassword = (char*)malloc(MAX_SIZE * sizeof(char));
  hipMalloc((void**) &cudaTestword, sizeof(char) * MAX_SIZE);
  hipMemcpy(cudaTestword, testword, sizeof(char) * MAX_SIZE, hipMemcpyHostToDevice);

  printf("Please enter a password:\n - cannot be over %d characters\n - can contain letters, numbers, and symbols\n - no spaces\n", MAX_SIZE - 1);
  scanf("%s", hostPassword);

  hipMemcpyToSymbol(HIP_SYMBOL(password), hostPassword, MAX_SIZE * sizeof(char));

  hipEventRecord(start);
  startCrackin<<<blocks, VARIATION>>>(cudaTestword);
  hipEventRecord(stop);

  hipMemcpy(testword, cudaTestword, sizeof(char) * MAX_SIZE, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  float runtime;
  hipEventElapsedTime(&runtime, start, stop);
  printf("It took %.6f milliseconds to guess the word %s\n", runtime, testword);
  free(testword);
  free(hostPassword);
  hipFree(cudaTestword);
}
